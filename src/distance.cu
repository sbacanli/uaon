

#include <hip/hip_runtime.h>
#include <math.h>
extern "C"
__global__ void relative(double x1, double y1, double x2, double y2,double* result )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
	double returned=0;
	double sum=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);
    returned=sqrt(sum);
	if (index == 0) result[0] = returned;
}

extern "C"
__global__ void real(double lat1,double lon1,double lat2,double lon2,double* result )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	double returned=0;
	
    double R = 6371; // Radius of the earth in km
    double dLat = (lat2-lat1)*3.14159265359/180;  // deg2rad below
    double dLon = (lon2-lon1)*3.14159265359/180; 
    double a = 
    		sin(dLat/2) * sin(dLat/2) +
    		cos((lat1*3.14159265359/180)) * cos((lat2*3.14159265359/180)) * 
    		sin(dLon/2) * sin(dLon/2); 
    double c = 2 * atan2(sqrt(a), sqrt(1-a)); 
    double d = R * c; // Distance in km
    returned=d*1000;
	
	if (index == 0) result[0] = returned;
}

/*
double deg2rad(double deg) {
	double pi=3.14159265359;
	return deg * (pi/180);
}
*/

