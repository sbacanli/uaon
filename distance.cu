

#include <hip/hip_runtime.h>
#include <math.h>
extern "C"
__global__ void relative(float x1, float y1, float x2, float y2,float* result )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
	float returned=0;
	float sum=(x1-x2)*(x1-x2)+(y1-y2)*(y1-y2);
    returned=(float)sqrt((float)sum);
	if (index == 0) result[0] = returned;
}

extern "C"
__global__ void real(float lat1,float lon1,float lat2,float lon2,float* result )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	float returned=0;
	
    float R = 6371; // Radius of the earth in km
    float dLat = (lat2-lat1)* (3.14159265359/180);  // deg2rad below
    float dLon = (lon2-lon1)* (3.14159265359/180); 
    float a = 
    		sinf(dLat/2) * sinf(dLat/2) +
    		cosf(lat1* (3.14159265359/180)) * cosf(lat2* (3.14159265359/180)) * 
    		sinf(dLon/2) * sinf(dLon/2); 
    float c = 2 * atan2f(sqrt(a), sqrt(1-a)); 
    float d = R * c; // Distance in km
    returned=d*1000;
	
	if (index == 0) result[0] = returned;
}
